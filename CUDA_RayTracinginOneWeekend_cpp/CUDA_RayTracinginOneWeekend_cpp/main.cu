#include "hip/hip_runtime.h"
//
//  main.cpp
//  CUDA_RayTracinginOneWeekend_cpp
//
//  Created by ��Ʋ� on 2021/3/8.
//

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <fstream>
#include "Vec3.h"
using namespace std;

// check cuda error
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        hipDeviceReset();
        exit(99);
    }
}

__global__ void render(Vec3* fb, int max_x, int max_y) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    fb[pixel_index] = Vec3(float(i) / max_x, float(j) / max_y, 0.2f);
}

int main() {
    ofstream fout("C:\\Users\\haobo\\Desktop\\output.ppm");
    if (fout.fail()) {
        cout << "open file output.ppm failed" << endl;
        return -1;
    }

    int image_width = 256;
    int image_height = 256;
    int tx = 8;
    int ty = 8;

    std::cerr << "Rendering a " << image_width << "x" << image_height << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = image_width * image_height;
    size_t fb_size = 3 * num_pixels * sizeof(float);

    // allocate FB
    Vec3* fb;
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    // Render our buffer
    dim3 blocks(image_width / tx + 1, image_height / ty + 1);
    dim3 threads(tx, ty);
    render << <blocks, threads >> > (fb, image_width, image_height);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Output FB as Image
    fout << "P3\n" << image_width << ' ' << image_height << "\n255\n";
    for (int j = image_height - 1; j >= 0; j--) {
        std::cerr << "\rScanlines remaining: " << j << ' ' << std::flush;
        for (int i = 0; i < image_width; i++) {
            size_t pixel_index = j * image_width + i;
            int ir = int(255.99 * fb[pixel_index].r());
            int ig = int(255.99 * fb[pixel_index].g());
            int ib = int(255.99 * fb[pixel_index].b());
            fout << ir << " " << ig << " " << ib << "\n";
        }
    }

    checkCudaErrors(hipFree(fb));

    std::cerr << "\nDone.\n";

    fout.close();

    return 0;
}