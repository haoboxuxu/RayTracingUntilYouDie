#include "hip/hip_runtime.h"
//
//  main.cpp
//  CUDA_RayTracinginOneWeekend_cpp
//
//  Created by ��Ʋ� on 2021/3/8.
//

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <fstream>
using namespace std;

// check cuda error
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        hipDeviceReset();
        exit(99);
    }
}

__global__ void render(float* fb, int max_x, int max_y) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x * 3 + i * 3;
    fb[pixel_index + 0] = float(i) / max_x;
    fb[pixel_index + 1] = float(j) / max_y;
    fb[pixel_index + 2] = 0.2;
}

int main() {
    ofstream fout("C:\\Users\\haobo\\Desktop\\output.ppm");
    if (fout.fail()) {
        cout << "open file output.ppm failed" << endl;
        return -1;
    }

    int image_width = 256;
    int image_height = 256;
    int tx = 8;
    int ty = 8;

    std::cerr << "Rendering a " << image_width << "x" << image_height << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = image_width * image_height;
    size_t fb_size = 3 * num_pixels * sizeof(float);

    // allocate FB
    float* fb;
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    // Render our buffer
    dim3 blocks(image_width / tx + 1, image_height / ty + 1);
    dim3 threads(tx, ty);
    render << <blocks, threads >> > (fb, image_width, image_height);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Output FB as Image
    fout << "P3\n" << image_width << ' ' << image_height << "\n255\n";
    for (int j = image_height - 1; j >= 0; j--) {
        std::cerr << "\rScanlines remaining: " << j << ' ' << std::flush;
        for (int i = 0; i < image_width; i++) {
            size_t pixel_index = j * 3 * image_width + i * 3;
            float r = fb[pixel_index + 0];
            float g = fb[pixel_index + 1];
            float b = fb[pixel_index + 2];
            int ir = int(255.99 * r);
            int ig = int(255.99 * g);
            int ib = int(255.99 * b);
            fout << ir << " " << ig << " " << ib << "\n";
        }
    }

    checkCudaErrors(hipFree(fb));

    std::cerr << "\nDone.\n";

    fout.close();

    return 0;
}