#include "hip/hip_runtime.h"
//
//  main.cpp
//  CUDA_RayTracinginOneWeekend_cpp
//
//  Created by ��Ʋ� on 2021/3/8.
//

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <fstream>
#include "Vec3.h"
#include "Ray.h"
#include "Sphere.h"
#include "Hitable.h"
#include "HitableList.h"
using namespace std;

// check cuda error
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        hipDeviceReset();
        exit(99);
    }
}

__device__ bool hit_sphere(const Point3& center, float radius, const Ray& r) {
    Vec3 oc = r.origin() - center;
    float a = dot(r.direction(), r.direction());
    float b = 2.0f * dot(oc, r.direction());
    float c = dot(oc, oc) - radius * radius;
    float discriminant = b * b - 4.0f * a * c;
    return (discriminant > 0.0f);
}

__device__ Vec3 color(const Ray& r, Hitable** world) {
    HitRecord rec;
    if ((*world)->hit(r, 0.0, FLT_MAX, rec)) {
        return 0.5f * Vec3(rec.normal.x() + 1.0f, rec.normal.y() + 1.0f, rec.normal.z() + 1.0f);
    } else {
        Vec3 unit_direction = unit_vector(r.direction());
        float t = 0.5f * (unit_direction.y() + 1.0f);
        return (1.0f - t) * Vec3(1.0, 1.0, 1.0) + t * Vec3(0.5, 0.7, 1.0);
    }
}

__global__ void render(Vec3* fb, int max_x, int max_y, Vec3 lower_left_corner, Vec3 horizontal, Vec3 vertical, Vec3 origin, Hitable** world) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    float u = float(i) / float(max_x);
    float v = float(j) / float(max_y);
    Ray r(origin, lower_left_corner + u * horizontal + v * vertical);
    fb[pixel_index] = color(r, world);
}

__global__ void create_world(Hitable** d_list, Hitable** d_world) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *(d_list) = new Sphere(Vec3(0, 0, -1), 0.5);
        *(d_list + 1) = new Sphere(Vec3(0, -100.5, -1), 100);
        *d_world = new HitableList(d_list, 2);
    }
}

__global__ void free_world(Hitable** d_list, Hitable** d_world) {
    delete* (d_list);
    delete* (d_list + 1);
    delete* d_world;
}

int main() {
    ofstream fout("C:\\Users\\haobo\\Desktop\\output.ppm");
    if (fout.fail()) {
        cout << "open file output.ppm failed" << endl;
        return -1;
    }

    // Image
    const auto aspect_ratio = 16.0 / 9.0;
    const int image_width = 400;
    const int image_height = static_cast<int>(image_width / aspect_ratio);
    int tx = 8;
    int ty = 8;

    // World
    Hitable** d_list;
    checkCudaErrors(hipMalloc((void**)&d_list, 2 * sizeof(Hitable*)));
    Hitable** d_world;
    checkCudaErrors(hipMalloc((void**)&d_world, sizeof(Hitable*)));
    create_world << <1, 1 >> > (d_list, d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Camera
    auto viewport_height = 2.0;
    auto viewport_width = aspect_ratio * viewport_height;
    auto focal_length = 1.0;
    auto origin = Point3(0, 0, 0);
    auto horizontal = Vec3(viewport_width, 0, 0);
    auto vertical = Vec3(0, viewport_height, 0);
    auto lower_left_corner = origin - horizontal / 2 - vertical / 2 - Vec3(0, 0, focal_length);

    std::cerr << "Rendering a " << image_width << "x" << image_height << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = image_width * image_height;
    size_t fb_size = 3 * num_pixels * sizeof(float);

    // allocate FB
    Vec3* fb;
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    // Render our buffer
    dim3 blocks(image_width / tx + 1, image_height / ty + 1);
    dim3 threads(tx, ty);
    render << <blocks, threads >> > (fb, image_width, image_height,
                                     lower_left_corner,
                                     horizontal,
                                     vertical,
                                     origin, d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Output FB as Image
    fout << "P3\n" << image_width << ' ' << image_height << "\n255\n";
    for (int j = image_height - 1; j >= 0; j--) {
        std::cerr << "\rScanlines remaining: " << j << ' ' << std::flush;
        for (int i = 0; i < image_width; i++) {
            size_t pixel_index = j * image_width + i;
            int ir = int(255.99 * fb[pixel_index].r());
            int ig = int(255.99 * fb[pixel_index].g());
            int ib = int(255.99 * fb[pixel_index].b());
            fout << ir << " " << ig << " " << ib << "\n";
        }
    }

    checkCudaErrors(hipDeviceSynchronize());
    free_world << <1, 1 >> > (d_list, d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(fb));

    std::cerr << "\nDone.\n";

    fout.close();

    return 0;
}