#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <algorithm>
#include <iostream>
#include <vector>
#include <cassert>
using std::begin;
using std::copy;
using std::cout;
using std::end;
using std::generate;
using std::vector;

// cuda kernel for vec add
__global__ void vectorAdd(int* a, int* b, int* c, int N) {
	// calculate global thread id
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (tid < N) c[tid] = a[tid] + b[tid];
}

// check results
void verify_results(int* a, int* b, int* c, int N) {
	for (int i = 0; i < N; i++) {
		assert(c[i] == a[i] + b[i]);
	}
	std::cout << "finished verify, no error\n";
}

int main() {
	// array size
	constexpr int N = 1 << 26;
	size_t bytes = sizeof(int) * N;

	// vectors cpu-side
	int* h_a, * h_b, * h_c;

	// alloc pinned memory
	hipHostMalloc(&h_a, bytes);
	hipHostMalloc(&h_b, bytes);
	hipHostMalloc(&h_c, bytes);

	// init nums in array
	for (int i = 0; i < N; i++) {
		h_a[i] = rand() % 100;
		h_b[i] = rand() % 100;
	}

	// alloc memory on device
	int* d_a, * d_b, * d_c;
	hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);
	hipMalloc(&d_c, bytes);

	// copy cpu->gpu
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

	// threads per cta
	int NUM_THREADS = 1 << 10;

	// cta per grid
	int NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;

	// run kernel on gpu
	vectorAdd << <NUM_BLOCKS, NUM_THREADS >> > (d_a, d_b, d_c, N);

	// copy sum gpu->cpu
	hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

	// check
	verify_results(h_a, h_b, h_c, N);

	// free
	hipHostFree(h_a);
	hipHostFree(h_b);
	hipHostFree(h_c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	std::cout << "COMPLETED SUCCESSFULLY\n";

	return 0;
}